#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "renderer/RayType.h"
#include "renderer/random.h"
#include "renderer/SubpathPRD.h"

using namespace optix;
using namespace ContextTest;

rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );


// <From OptiX path_trace sample>
// Create ONB from normalaized vector
static __device__ __inline__ void createONB( 
	const optix::float3& n, optix::float3& U, optix::float3& V)
{
	using namespace optix;

	U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
	if ( dot(U, U) < 1.e-3f )
		U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
	U = normalize( U );
	V = cross( n, U );
}


float3 __device__ __inline__ sampleHemisphereCosOptix(float3 normal, float2 rnd)
{
	float3 p;
	cosine_sample_hemisphere(rnd.x, rnd.y, p);
	float3 v1, v2;
	createONB(normal, v1, v2);
	return v1 * p.x + v2 * p.y + normal * p.z;  
}
// </From OptiX path_trace sample>


// NOTE:
// All fail case due setting cosine sampled direction were tested with all rtPrintf statements
// in the generation program commented out
RT_PROGRAM void closestHit()
{
	lightPrd.depth++;

	//if (lightPrd.depth == 2)                                // doesn't prevent crash on second hit
	//{                                                       // when using #1
	//	lightPrd.done = 1;
	//	return;
	//}

	float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
	float3 hitPoint = ray.origin + tHit*ray.direction;

	// Kind of Russian Roulette
	if (0.7f < rnd(lightPrd.seed))
	{
		lightPrd.done = 1;
		return;
	}

	float2 bsdfSample = make_float2(rnd(lightPrd.seed),rnd(lightPrd.seed));
	float3 dir = sampleHemisphereCosOptix(worldShadingNormal, bsdfSample); // --> #1 doesn't work

	//dir = normalize(2*worldShadingNormal + ray.direction);  // --> #2 works (computation in #1 can be left uncommented)
	//dir = -ray.direction;                                   // --> #3 works (computation in #1 can be left uncommented)
  
	//if (1 < lightPrd.depth)                                 // #1 still causes crash - this shows that crash occurs because of setting
	//    dir = -ray.direction;                               // lightsPrd.direction to cosine sampled direction on first hit

	lightPrd.direction = normalize(dir);     
	lightPrd.origin = hitPoint;

	// #1 doesn't crash if code below uncommented (stop on first hit)
	//if (lightPrd.depth == 1)	// #1 crashes if condition is depth == 2
	//{							// even though the new direction is never used
	//	lightPrd.done = 1;		// to trace a ray
	//	return;
	//}
}


// THIS WORKS with generatorRecursive
RT_PROGRAM void closestHitRecursive()
{
	lightPrd.depth++;
	float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
	float3 hitPoint = ray.origin + tHit*ray.direction;

	// Kind of Russian Roulette
	if (0.7f < rnd(lightPrd.seed))
	{
		lightPrd.done = 1;
		return;
	}

	float2 bsdfSample = make_float2(rnd(lightPrd.seed),rnd(lightPrd.seed));
	float3 dir = sampleHemisphereCosOptix(worldShadingNormal, bsdfSample);

	lightPrd.direction = normalize(dir);     
	lightPrd.origin = hitPoint;

	Ray newRay = Ray(lightPrd.origin, lightPrd.direction, RayType::LIGHT_VCM, 0.0001, RT_DEFAULT_MAX );
	rtTrace( sceneRootObject, newRay, lightPrd );
}