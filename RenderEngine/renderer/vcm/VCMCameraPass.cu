#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/Camera.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "renderer/vcm/PathVertex.h"
#include "renderer/vcm/SubpathPRD.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(Camera, camera, , );
rtBuffer<Light, 1> lights;
rtBuffer<float3, 2> outputBuffer;
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
//rtDeclareVariable(Sphere, sceneBoundingSphere, , );

// VCM
rtDeclareVariable(uint2, pixelSizeFactor, , );
rtDeclareVariable(float, vcmMisVcWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(float, vcmMisVmWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(float, vcmLightSubpathCount, , ); // vmarz TODO set


RT_PROGRAM void cameraPass()
{
    SubpathPRD cameraPrd;
    cameraPrd.randomState = randomStates[launchIndex];
    cameraPrd.throughput = make_float3(1.0f);
    cameraPrd.depth = 0u;
    cameraPrd.done = 0u;    

    float2 screen = make_float2( outputBuffer.size() );
    float2 sample = getRandomUniformFloat2(&cameraPrd.randomState);             // jitter pixel pos
    float2 d = ( make_float2(launchIndex) + sample ) / screen * 2.0f - 1.0f;    // vmarz: map pixel pos to [-1,1]
    
    float3 rayOrigin = camera.eye;
    float3 rayDirection = normalize(d.x*camera.camera_u + d.y*camera.camera_v + camera.lookdir);
    //modifyRayForDepthOfField(camera, rayOrigin, rayDirection, radiancePrd.randomState);     // vmarz TODO add ?
    Ray cameraRay = Ray(rayOrigin, rayDirection, RayType::CAMERA_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );
    cameraPrd.origin = rayOrigin;
    cameraPrd.direction = rayDirection;

    // pdf conversion factor from area on image plane to solid angle on ray
    float cosAtCamera = dot(normalize(camera.lookdir), rayDirection);
    float imagePointToCameraDist = length(camera.lookdir) / cosAtCamera;
    float imageToSolidAngleFactor = sqr(imagePointToCameraDist) / cosAtCamera;

    float pixelArea = pixelSizeFactor.x * camera.imagePlaneSize.x * pixelSizeFactor.x * camera.imagePlaneSize.y;
    float cameraPdfW = (1.0f/pixelArea) * imageToSolidAngleFactor;

    //cameraPrd.specularPath = 1; // vmarz TODO ?

    cameraPrd.dVC = .0f;
    cameraPrd.dVM = .0f;
    //cameraPrd.dVCM = vcmMis(vcmLightSubpathCount / cameraPdfW);

    // Trace    
    for (int i=0;;i++)
    {
        //OPTIX_DEBUG_PRINT(cameraPrd.depth, "G %d - tra dir %f %f %f\n",
        //    i, cameraRay.direction.x, cameraRay.direction.y, cameraRay.direction.z);
        rtTrace( sceneRootObject, cameraRay, cameraPrd );
        
        // sample direct lightning

        // vertext connection

        // vertex merging

        if (cameraPrd.done)
        {
            //OPTIX_DEBUG_PRINT(cameraPrd.depth, "Stop trace \n");
            break;
        }

        // sample new dir

        cameraRay.origin = cameraPrd.origin;
        cameraRay.direction = cameraPrd.direction;

        //OPTIX_DEBUG_PRINT(cameraPrd.depth, "G %d - new org %f %f %f\n", i, cameraRay.origin.x, cameraRay.origin.y, cameraRay.origin.z);
        //OPTIX_DEBUG_PRINT(cameraPrd.depth, "G %d - new dir %f %f %f\n", i, cameraRay.direction.x, cameraRay.direction.y, cameraRay.direction.z);
    }

    randomStates[launchIndex] = cameraPrd.randomState;
}


rtDeclareVariable(SubpathPRD, cameraPrd, rtPayload, );
RT_PROGRAM void miss()
{
    cameraPrd.done = 1;
    OPTIX_DEBUG_PRINT(cameraPrd.depth, "Miss\n");
    //rtPrintf("%d %d: MISS depth %d ndir %f %f %f\n", launchIndex.x, launchIndex.y, cameraPrd.depth,
    //            cameraPrd.direction.x, cameraPrd.direction.y, cameraPrd.direction.z);
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    rtPrintf("Exception VCM Camera ray! d: %d\n", cameraPrd.depth);
    rtPrintExceptionDetails();
    cameraPrd.throughput = make_float3(1,0,0);
}
