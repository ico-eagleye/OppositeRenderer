#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#define OPTIX_PRINTFID_DISABLE
#define OPTIX_PRINTFI_DISABLE
#define OPTIX_PRINTFIALL_DISABLE

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/Camera.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "renderer/vcm/LightVertex.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/vcm.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(Camera, camera, , );
rtBuffer<Light, 1> lights;
rtBuffer<float3, 2> outputBuffer;                   // TODO change to float4
rtDeclareVariable(uint, localIterationNumber, , );
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
//rtDeclareVariable(Sphere, sceneBoundingSphere, , );

// VCM
rtDeclareVariable(float2, pixelSizeFactor, , );
rtDeclareVariable(float, vcmMisVcWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(float, vcmMisVmWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(uint, vcmLightSubpathCount, , ); // vmarz TODO set

static __device__ __inline__ float3 averageInNewRadiance(const float3 newRadiance, const float3 oldRadiance, const float localIterationNumber)
{
    if(localIterationNumber >= 1)
    {
        return oldRadiance + (newRadiance-oldRadiance)/(localIterationNumber+1);
    }
    else
    {
        return newRadiance;
    }
}


RT_PROGRAM void cameraPass()
{
    //if (launchIndex.x != 0 || launchIndex.y != 0) return;
    SubpathPRD cameraPrd;
    cameraPrd.randomState = randomStates[launchIndex];
    cameraPrd.throughput = make_float3(1.0f);
    cameraPrd.color = make_float3(0.0f);
    cameraPrd.depth = 0;
    cameraPrd.done = 0;
    cameraPrd.dVC = 0;
    cameraPrd.dVM = 0;
    cameraPrd.dVCM = 0;
    cameraPrd.launchIndex = launchIndex;

    float2 screen = make_float2( outputBuffer.size() );
    float2 sample = getRandomUniformFloat2(&cameraPrd.randomState);             // jitter pixel pos
    float2 d = ( make_float2(launchIndex) + sample ) / screen * 2.0f - 1.0f;    // vmarz: map pixel pos to [-1,1]
    
    cameraPrd.origin = camera.eye;
    cameraPrd.direction = normalize(d.x*camera.camera_u + d.y*camera.camera_v + camera.lookdir);
    //modifyRayForDepthOfField(camera, rayOrigin, rayDirection, radiancePrd.randomState);     // vmarz TODO add ?
    Ray cameraRay = Ray(cameraPrd.origin, cameraPrd.direction, RayType::CAMERA_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );

    initCameraPayload(cameraPrd, camera, pixelSizeFactor, vcmLightSubpathCount);
    OPTIX_PRINTFI(0, "Gen C - start - dVCM %f\n", cameraPrd.dVCM);

    // Trace    
    for (int i=0;;i++)
    {
        //OPTIX_PRINTFI(cameraPrd.depth, "G %d - tra dir %f %f %f\n",
        //    i, cameraRay.direction.x, cameraRay.direction.y, cameraRay.direction.z);
        rtTrace( sceneRootObject, cameraRay, cameraPrd );
        
        // sample direct lightning

        // vertext connection

        // vertex merging

        if (cameraPrd.done)
        {
            //OPTIX_PRINTFI(cameraPrd.depth, "Stop trace \n");
            break;
        }

        // sample new dir

        cameraRay.origin = cameraPrd.origin;
        cameraRay.direction = cameraPrd.direction;

        //OPTIX_PRINTFI(cameraPrd.depth, "G %d - new org %f %f %f\n", i, cameraRay.origin.x, cameraRay.origin.y, cameraRay.origin.z);
        //OPTIX_PRINTFI(cameraPrd.depth, "G %d - new dir %f %f %f\n", i, cameraRay.direction.x, cameraRay.direction.y, cameraRay.direction.z);
    }

    float3 avgColor = averageInNewRadiance(cameraPrd.color, outputBuffer[launchIndex], localIterationNumber);
    OPTIX_PRINTFI(cameraPrd.depth, "Gen C - DONE colr % 14f % 14f % 14f\n", cameraPrd.color.x, cameraPrd.color.y, cameraPrd.color.z);
    OPTIX_PRINTFI(cameraPrd.depth, "             avg  % 14f % 14f % 14f\n", avgColor.x, avgColor.y, avgColor.z);

    //OPTIX_PRINTF("%d , %d - d %d - iter %d prd.color %f %f %f avColor %f %f %f\n", 
    //    launchIndex.x, launchIndex.y, cameraPrd.depth, localIterationNumber,
    //    cameraPrd.color.x, cameraPrd.color.y, cameraPrd.color.z, avgColor.x, avgColor.y, avgColor.z);

    outputBuffer[launchIndex] = avgColor;
    randomStates[launchIndex] = cameraPrd.randomState;
}


rtDeclareVariable(SubpathPRD, cameraPrd, rtPayload, );
RT_PROGRAM void miss()
{
    cameraPrd.done = 1;
    //OPTIX_PRINTFI(cameraPrd.depth, "Miss\n");
    OPTIX_PRINTFI(cameraPrd.depth, "Gen C -      MISS dirW % 14f % 14f % 14f           from % 14f % 14f % 14f \n",
                      cameraPrd.direction.x, cameraPrd.direction.y, cameraPrd.direction.z,
                      cameraPrd.origin.x, cameraPrd.origin.y, cameraPrd.origin.z);
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    rtPrintf("Exception VCM Camera ray! d: %d\n", cameraPrd.depth);
    rtPrintExceptionDetails();
    cameraPrd.throughput = make_float3(1,0,0);
}
