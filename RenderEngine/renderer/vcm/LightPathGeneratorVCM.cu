#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/ShadowPRD.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "renderer/ppm/Photon.h"
#include "renderer/ppm/PhotonPRD.h"
#include "math/Sphere.h"

#include "renderer/vcm/PathVertex.h"
#include "renderer/vcm/SubpathPRD.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Photon, 1> photons;
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(uint, photonLaunchWidth, , );
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

//optix::float3 __inline __device__ lightEmit(const Light & aLight, RandomState & aRandomState,
//											float3 & oPosition, float3 & oDirection, float & oEmissionPdfW,
//											float & oDirectPdfA, float & oCosThetaLight)
//struct SubpathPRD
//{
//    optix::float3 origin;
//	optix::float3 direction;
//	optix::float3 throughput;
//    optix::uint depth;
//    RandomState randomState;
//	float dVCM;
//	float dVC;
//	float dVM;
//	//uint  mIsFiniteLight :  1; // Just generate by finite light
//    //uint  mSpecularPath  :  1; // All scattering events so far were specular
//};


RT_PROGRAM void generator()
{
	SubpathPRD lightPRD;
	lightPRD.depth = 0;
	lightPRD.randomState = randomStates[launchIndex];
	lightPRD.dVC = 0;
	lightPRD.dVM = 0;
	lightPRD.dVCM = 0;

	// vmarz?: pick based on light power?
	int lightIndex = 0;
	if (1 < lights.size())
	{
		float sample = getRandomUniformFloat(&lightPRD.randomState);
		lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
	}

	const Light light = lights[lightIndex];
	const float inverseLightPickPdf = lights.size();

	float3 rayOrigin, rayDirection;
	float emissionPdfW, directPdfW, cosAtLight;
	lightPRD.throughput = lightEmit(light, lightPRD.randomState, rayOrigin, rayDirection, emissionPdfW, directPdfW, cosAtLight);
	// vmarz?: do something similar as done for photon, emit towards scene when light far from scene?
	// check if photons normally missing the scene accounted for?

	emissionPdfW *= inverseLightPickPdf;
	directPdfW *= inverseLightPickPdf;

	lightPRD.throughput /= emissionPdfW;
	//lightPRD.isFinite = isDelta.isFinite ... vmarz?

	// e.g. if not delta ligth
	if (!light.isDelta)
	{
		const float usedCosLight = light.isFinite ? cosAtLight : 1.f;
		lightPRD.dVC = Mis(usedCosLight / emissionPdfW);
	}

	//lightPRD.dVM = lightPRD.dVC * mMisVcWeightFactor; // vmarz: TODO

	Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, 0.0001, RT_DEFAULT_MAX );
	rtTrace( sceneRootObject, lightRay, lightPRD );

	randomStates[launchIndex] = lightPRD.randomState;

#if ENABLE_RENDER_DEBUG_OUTPUT
	debugPhotonPathLengthBuffer[launchIndex] = lightPRD.depth;
#endif
}

rtDeclareVariable(SubpathPRD, lightPRD, rtPayload, );
RT_PROGRAM void miss()
{
    OPTIX_DEBUG_PRINT(lightPRD.depth, "Light ray missed geometry.\n");
}

//
// Exception handler program
//
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    printf("Exception Light ray!\n");
    lightPRD.throughput = make_float3(0,0,0);
}