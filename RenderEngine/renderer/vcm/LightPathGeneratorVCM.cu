#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/ShadowPRD.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "math/Sphere.h"

#include "renderer/vcm/PathVertex.h"
#include "renderer/vcm/SubpathPRD.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

// VCM
rtDeclareVariable(float, misVcWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(float, misVmWeightFactor, , ); // vmarz TODO set

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

rtBuffer<uint, 2> lightVertexCountBuffer;
rtBuffer<uint, 2> dbgNoMissHitStops;

RT_PROGRAM void generatorEstimate()
{
    SubpathPRD lightPrd;
    lightPrd.depth = 0;
    lightPrd.keepTracing = 0;
    lightPrd.done = 0;
    lightPrd.randomState = randomStates[launchIndex];
    lightPrd.dVC = 0;
    lightPrd.dVM = 0;
    lightPrd.dVCM = 0;
    lightVertexCountBuffer[launchIndex] = 0u;
    dbgNoMissHitStops[launchIndex] = 0u;

    // vmarz TODO: pick based on light power
    int lightIndex = 0;
    if (1 < lights.size())
    {
        float sample = getRandomUniformFloat(&lightPrd.randomState);
        lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
    }

    const Light light = lights[lightIndex];
    const float inverseLightPickPdf = lights.size();

    float3 rayOrigin;
    float3 rayDirection;
    float emissionPdfW;
    float directPdfW;
    float cosAtLight;
    lightPrd.throughput = lightEmit(light, lightPrd.randomState, rayOrigin, rayDirection, emissionPdfW, directPdfW, cosAtLight);
    // vmarz?: do something similar as done for photon, emit towards scene when light far from scene?
    // check if photons normally missing the scene accounted for?

    // Set init data
    emissionPdfW *= inverseLightPickPdf;
    directPdfW *= inverseLightPickPdf;

    lightPrd.throughput /= emissionPdfW;
    //lightPrd.isFinite = isDelta.isFinite ... vmarz?

    lightPrd.dVCM = Mis(directPdfW / emissionPdfW);

    // e.g. if not delta ligth
    //if (!light.isDelta)
    //{
    //	const float usedCosLight = light.isFinite ? cosAtLight : 1.f;
    //	lightPrd.dVC = Mis(usedCosLight / emissionPdfW);
    //}

    lightPrd.dVM = lightPrd.dVC * misVcWeightFactor;

    //dbg
    //rayOrigin = make_float3( 343.0f, 548.7999f, 227.0f);
    //rayDirection = make_float3( .0f, -1.0f, .0f);

    // Trace
    Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );
    
    for (int i=0;;i++)
    {
        lightPrd.keepTracing = 0; // any hit sets this to one if continuing, done this way since rtTrace sometimes
                                  // doesn't result in miss or anythit being called
                                  // https://devtalk.nvidia.com/default/topic/754670/optix/rttrace-occasionally-results-in-nothing-no-call-to-any-hit-miss-or-exception-program-/
        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - tra dir %f %f %f\n",
            i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

        if (!lightPrd.keepTracing)
        {
            if (!lightPrd.done)
                dbgNoMissHitStops[launchIndex] = 1;

            OPTIX_DEBUG_PRINT(lightPrd.depth, "Stop trace \n");
            break;
        }

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;

        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - new org %f %f %f\n", i, lightRay.origin.x, lightRay.origin.y, lightRay.origin.z);
        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - new dir %f %f %f\n", i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);

        //if (lightPrd.depth == 2)
        //{
        //    //rtPrintf("%d %d: depth %d prd max - ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
        //    //    lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
        //    break;
        //}

        //if (i == 3)
        //{
        //    OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - itr max - ndir %f %f %f\n",
        //        i, lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);

        //    //rtPrintf("%d %d: depth %d iter max - ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
        //    //    lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
        //    break;
        //}
    }

    randomStates[launchIndex] = lightPrd.randomState;
}



RT_PROGRAM void generator()
{
    SubpathPRD lightPrd;
    lightPrd.depth = 0;
    lightPrd.keepTracing = 0;
    lightPrd.done = 0;
    lightPrd.randomState = randomStates[launchIndex];
    lightPrd.dVC = 0;
    lightPrd.dVM = 0;
    lightPrd.dVCM = 0;
    lightVertexCountBuffer[launchIndex] = 0u;
    dbgNoMissHitStops[launchIndex] = 0u;

    // vmarz TODO: pick based on light power
    int lightIndex = 0;
    if (1 < lights.size())
    {
        float sample = getRandomUniformFloat(&lightPrd.randomState);
        lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
    }

    const Light light = lights[lightIndex];
    const float inverseLightPickPdf = lights.size();

    float3 rayOrigin;
    float3 rayDirection;
    float emissionPdfW;
    float directPdfW;
    float cosAtLight;
    lightPrd.throughput = lightEmit(light, lightPrd.randomState, rayOrigin, rayDirection, emissionPdfW, directPdfW, cosAtLight);
    // vmarz?: do something similar as done for photon, emit towards scene when light far from scene?
    // check if photons normally missing the scene accounted for?

    // Set init data
    emissionPdfW *= inverseLightPickPdf;
    directPdfW *= inverseLightPickPdf;

    lightPrd.throughput /= emissionPdfW;
    //lightPrd.isFinite = isDelta.isFinite ... vmarz?

    lightPrd.dVCM = Mis(directPdfW / emissionPdfW);

    // e.g. if not delta ligth
    //if (!light.isDelta)
    //{
    //	const float usedCosLight = light.isFinite ? cosAtLight : 1.f;
    //	lightPrd.dVC = Mis(usedCosLight / emissionPdfW);
    //}

    lightPrd.dVM = lightPrd.dVC * misVcWeightFactor;

    //dbg
    //rayOrigin = make_float3( 343.0f, 548.7999f, 227.0f);
    //rayDirection = make_float3( .0f, -1.0f, .0f);

    // Trace
    Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );
    
    for (int i=0;;i++)
    {
        lightPrd.keepTracing = 0; // any hit sets this to one if continuing, done this way since rtTrace sometimes
                                  // doesn't result in miss or anythit being called
                                  // https://devtalk.nvidia.com/default/topic/754670/optix/rttrace-occasionally-results-in-nothing-no-call-to-any-hit-miss-or-exception-program-/
        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - tra dir %f %f %f\n",
            i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

        if (!lightPrd.keepTracing)
        {
            if (!lightPrd.done)
                dbgNoMissHitStops[launchIndex] = 1;

            OPTIX_DEBUG_PRINT(lightPrd.depth, "Stop trace \n");
            break;
        }

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;

        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - new org %f %f %f\n", i, lightRay.origin.x, lightRay.origin.y, lightRay.origin.z);
        OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - new dir %f %f %f\n", i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);

        //if (lightPrd.depth == 2)
        //{
        //    //rtPrintf("%d %d: depth %d prd max - ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
        //    //    lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
        //    break;
        //}

        if (i == 30)
        {
            OPTIX_DEBUG_PRINT(lightPrd.depth, "G %d - itr max - ndir %f %f %f\n",
                i, lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);

            //rtPrintf("%d %d: depth %d iter max - ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
            //    lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
            break;
        }
    }

    randomStates[launchIndex] = lightPrd.randomState;
}




RT_PROGRAM void generatorEstimateDbg()
{
    SubpathPRD lightPrd;
    lightPrd.depth = 0;
    lightPrd.done = 0;
    lightPrd.keepTracing = 0;
    lightPrd.randomState = randomStates[launchIndex]; // hiprand states
    lightPrd.seed = tea<16>(720u*launchIndex.y+launchIndex.x, 1u);
    //lightVertexCountBuffer[launchIndex] = 0u;
    dbgNoMissHitStops[launchIndex] = 0u;

    float3 rayOrigin = make_float3( 343.0f, 548.0f, 227.0f);
    float3 rayDirection = make_float3( .0f, -1.0f, .0f);
    Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, 0.0001, RT_DEFAULT_MAX );

    for (int i=0;;i++)
    {
        lightPrd.keepTracing = 0;
        //OPTIX_DEBUG_PRINT(lightPrd.depth, " dir %.2f %.2f %.2f\n",
        //    lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

        if (!lightPrd.keepTracing) 
        {
            if (!lightPrd.done)
                dbgNoMissHitStops[launchIndex] = 1;
            //lightPrd.done += a;
            //OPTIX_DEBUG_PRINT(lightPrd.depth, " done\n");
            break;
        }

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;
        //OPTIX_DEBUG_PRINT(lightPrd.depth, "Gen - new org %f %f %f\n", lightRay.origin.x, lightRay.origin.y, lightRay.origin.z);
        //OPTIX_DEBUG_PRINT(lightPrd.depth, "Gen - new org %f %f %f\n", lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
    }

    randomStates[launchIndex] = lightPrd.randomState;
}


rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );
RT_PROGRAM void miss()
{
    lightPrd.done = 1;
    OPTIX_DEBUG_PRINT(lightPrd.depth, "Miss\n");
    //rtPrintf("%d %d: MISS depth %d ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
    //            lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    rtPrintf("Exception Light ray! d: %d\n", lightPrd.depth);
    rtPrintExceptionDetails();
    lightPrd.throughput = make_float3(0,0,0);
}
