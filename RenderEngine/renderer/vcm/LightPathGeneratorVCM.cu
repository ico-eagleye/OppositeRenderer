#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "math/Sphere.h"

#include "renderer/vcm/LightVertex.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/vcm.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

rtDeclareVariable(uint, lightVertexCountEstimatePass, , );
rtDeclareVariable(float, misVcWeightFactor, , ); // 1/etaVCM
//rtDeclareVariable(float, misVmWeightFactor, , ); // etaVCM

rtBuffer<uint, 2> lightVertexCountBuffer;

RT_PROGRAM void lightPass()
{
    SubpathPRD lightPrd;
    lightPrd.depth = 0;
    lightPrd.done = 0;
    lightPrd.dVC = 0;
    lightPrd.dVM = 0;
    lightPrd.dVCM = 0;
    lightPrd.randomState = randomStates[launchIndex];
    if (lightVertexCountEstimatePass)
        lightVertexCountBuffer[launchIndex] = 0u;

    // vmarz TODO: pick based on light power
    int lightIndex = 0;
    if (1 < lights.size())
    {
        float sample = getRandomUniformFloat(&lightPrd.randomState);
        lightIndex = intmin((int)(sample*lights.size()), int(lights.size()-1));
    }

    const Light light = lights[lightIndex];
    const float inverseLightPickPdf = lights.size();
    const float lightPickPdf = 1.f / lights.size();

    // Initialize payload and ray
    initLightPayload(lightPrd, light, lightPickPdf, misVcWeightFactor);
    Ray lightRay = Ray(lightPrd.origin, lightPrd.direction, RayType::LIGHT_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );

    for (int i=0;;i++)
    {
        //OPTIX_PRINTFI(lightPrd.depth, "G %d - tra dir %f %f %f\n",
        //    i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

        if (lightPrd.done)
        {
            //OPTIX_PRINTFI(lightPrd.depth, "Stop trace \n");
            break;
        }

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;

        //OPTIX_PRINTFI(lightPrd.depth, "G %d - new org %f %f %f\n", i, lightRay.origin.x, lightRay.origin.y, lightRay.origin.z);
        //OPTIX_PRINTFI(lightPrd.depth, "G %d - new dir %f %f %f\n", i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
    }

    randomStates[launchIndex] = lightPrd.randomState;
}



rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );
RT_PROGRAM void miss()
{
    lightPrd.done = 1;
    //OPTIX_PRINTFI(lightPrd.depth, "Miss\n");
    //OPTIX_PRINTFI(lightPrd.depth, "%d %d: MISS depth %d ndir %f %f %f\n", launchIndex.x, launchIndex.y, lightPrd.depth,
    //            lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    rtPrintf("Exception Light ray! d: %d\n", lightPrd.depth);
    rtPrintExceptionDetails();
    lightPrd.throughput = make_float3(0,0,0);
}