#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

//#define OPTIX_PRINTFID_DISABLE
//#define OPTIX_PRINTFI_DISABLE
//#define OPTIX_PRINTFIALL_DISABLE

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "math/Sphere.h"

#include "renderer/vcm/LightVertex.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/vcm.h"
#include "renderer/vcm/config_vcm.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

rtDeclareVariable(uint, lightVertexCountEstimatePass, , );
rtDeclareVariable(float, misVcWeightFactor, , ); // 1/etaVCM
//rtDeclareVariable(float, misVmWeightFactor, , ); // etaVCM

rtBuffer<uint, 2> lightSubpathLengthBuffer;
rtBuffer<uint, 3> lightSubpathVertexIndexBuffer;
rtBuffer<LightVertex> lightVertexBuffer;
rtDeclareVariable(float, vertexPickPdf, , );

RT_PROGRAM void lightPass()
{
    SubpathPRD lightPrd;
    lightPrd.launchIndex = launchIndex;
    lightPrd.throughput = make_float3(1.f);
    lightPrd.depth = 0.f;
    lightPrd.done = 0.f;
    lightPrd.dVC = 0.f;
    lightPrd.dVM = 0.f;
    lightPrd.dVCM = 0.f;
    lightPrd.randomState = randomStates[launchIndex];
    lightSubpathLengthBuffer[launchIndex] = 0u; // prob here?

    if (lightVertexCountEstimatePass)
    {
        OPTIX_PRINTFI(0, "GenCL - LIGHT ESTIMATE PASS -----------------------------------------------------------------\n");
    }
    else
        OPTIX_PRINTFI(0, "GenCL - LIGHT STORE PASS --------------------------------------------------------------------\n");

    // vmarz TODO: pick based on light power
    int lightIndex = 0;
    if (1 < lights.size())
    {
        float sample = getRandomUniformFloat(&lightPrd.randomState);
        lightIndex = intmin((int)(sample*lights.size()), int(lights.size()-1));
    }

    const Light light = lights[lightIndex];
    const float inverseLightPickPdf = lights.size();
    const float lightPickPdf = 1.f / lights.size();

    float *vertPickPdfPtr = NULL;
#if VCM_UNIFORM_VERTEX_SAMPLING
    vertPickPdfPtr = &vertexPickPdf;
#endif

    // Initialize payload and ray
    initLightPayload(lightPrd, light, lightPickPdf, misVcWeightFactor, vertPickPdfPtr);
    Ray lightRay = Ray(lightPrd.origin, lightPrd.direction, RayType::LIGHT_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX );

    for (int i=0;;i++)
    {
        //OPTIX_PRINTFI(lightPrd.depth, "G %d - tra dir %f %f %f\n",
        //    i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

        if (lightPrd.done)
        {
            OPTIX_PRINTFI(lightPrd.depth, "GenCL - DONE LIGHT RAY\n\n");
            break;
        }

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;

        //OPTIX_PRINTFI(lightPrd.depth, "G %d - new org %f %f %f\n", i, lightRay.origin.x, lightRay.origin.y, lightRay.origin.z);
        //OPTIX_PRINTFI(lightPrd.depth, "G %d - new dir %f %f %f\n", i, lightRay.direction.x, lightRay.direction.y, lightRay.direction.z);
    }

    randomStates[launchIndex] = lightPrd.randomState;
    lightSubpathLengthBuffer[launchIndex] = lightPrd.depth;
}



rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );
RT_PROGRAM void miss()
{
    lightPrd.done = 1;
    //OPTIX_PRINTFI(lightPrd.depth, "Miss\n");
    OPTIX_PRINTFI(lightPrd.depth, "GenCL -       MISS dirW % 14f % 14f % 14f           from % 14f % 14f % 14f \n",
                      lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z,
                      lightPrd.origin.x, lightPrd.origin.y, lightPrd.origin.z);
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    rtPrintf("Exception Light ray! d: %d\n", lightPrd.depth);
    rtPrintExceptionDetails();
    lightPrd.throughput = make_float3(0,0,0);
}