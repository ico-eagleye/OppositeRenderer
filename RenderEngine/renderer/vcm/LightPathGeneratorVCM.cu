#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/ShadowPRD.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/light.h"
#include "math/Sphere.h"

#include "renderer/vcm/PathVertex.h"
#include "renderer/vcm/SubpathPRD.h"


using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

// VCM
rtDeclareVariable(float, misVcWeightFactor, , ); // vmarz TODO set
rtDeclareVariable(float, misVmWeightFactor, , ); // vmarz TODO set

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

//optix::float3 __inline __device__ lightEmit(const Light & aLight, RandomState & aRandomState,
//											float3 & oPosition, float3 & oDirection, float & oEmissionPdfW,
//											float & oDirectPdfA, float & oCosThetaLight)
//struct SubpathPRD
//{
//    optix::float3 origin;
//	optix::float3 direction;
//	optix::float3 throughput;
//    optix::uint depth;
//    RandomState randomState;
//	float dVCM;
//	float dVC;
//	float dVM;
//	//uint  mIsFiniteLight :  1; // Just generate by finite light
//    //uint  mSpecularPath  :  1; // All scattering events so far were specular
//};

rtBuffer<ushort, 2> lightVertexCountBuffer;

RT_PROGRAM void generator()
{
	SubpathPRD lightPrd;
	//lightPrd.depth = 0;
    lightPrd.done = 0;
	//lightPrd.randomState = randomStates[launchIndex];
	//lightPrd.dVC = 0;
	//lightPrd.dVM = 0;
	//lightPrd.dVCM = 0;
    //lightVertexCountBuffer[launchIndex] = lightPrd.depth;

	// vmarz?: pick based on light power?
	//int lightIndex = 0;
	//if (1 < lights.size())
	//{
	//	float sample = getRandomUniformFloat(&lightPrd.randomState);
	//	lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
	//}

	//const Light light = lights[lightIndex];
	//const float inverseLightPickPdf = lights.size();

    const Light light = lights[0];
	float3 rayOrigin = make_float3( 343.0f, 548.7999f, 227.0f);
	float3 rayDirection = make_float3( .0f, -1.0f, .0f);
	//float emissionPdfW;
	//float directPdfW;
	//float cosAtLight;
	//lightPrd.throughput = lightEmit(light, lightPrd.randomState, rayOrigin, rayDirection, emissionPdfW, directPdfW, cosAtLight);
	//// vmarz?: do something similar as done for photon, emit towards scene when light far from scene?
	//// check if photons normally missing the scene accounted for?
	//
	//// Set init data
	//emissionPdfW *= inverseLightPickPdf;
	//directPdfW *= inverseLightPickPdf;

	//lightPrd.throughput /= emissionPdfW;
	//lightPrd.isFinite = isDelta.isFinite ... vmarz?

	//lightPrd.dVCM = Mis(directPdfW / emissionPdfW);

	// e.g. if not delta ligth
	//if (!light.isDelta)
	//{
	//	const float usedCosLight = light.isFinite ? cosAtLight : 1.f;
	//	lightPrd.dVC = Mis(usedCosLight / emissionPdfW);
	//}

	//lightPrd.dVM = lightPrd.dVC * misVcWeightFactor;

	// Trace
	Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, 0.0001, RT_DEFAULT_MAX );
	
    for (int i=0;;i++)
	{
        if ((launchIndex.x + launchIndex.y) != 0)
        {
            printf("Gen %d - idx %d,%d - break\n", i, launchIndex.x, launchIndex.y);
            break;
        }
        
        printf("Gen %d - idx %d,%d - Dir %f %f %f\n", i, launchIndex.x, launchIndex.y, 
            rayDirection.x, rayDirection.y, rayDirection.z);
        rtTrace( sceneRootObject, lightRay, lightPrd );

		if (lightPrd.done) 
        {
            printf("Gen %d - idx %d,%d - break\n", i, launchIndex.x, launchIndex.y);
            break;
        }
        //else
        //    lightPrd.done = 1;

        lightRay.origin = lightPrd.origin;
        lightRay.direction = lightPrd.direction;
        printf("Gen %d - idx %d,%d - isdone %d \n", i, launchIndex.x, launchIndex.y, i, lightPrd.done);
	}

	randomStates[launchIndex] = lightPrd.randomState;
    printf("Done idx %d,%d \n", launchIndex.x, launchIndex.y);
}


//RT_PROGRAM void generator()
//{
//	SubpathPRD lightPrd;
//	lightPrd.depth = 0;
//	lightPrd.randomState = randomStates[launchIndex];
//	lightPrd.dVC = 0;
//	lightPrd.dVM = 0;
//	lightPrd.dVCM = 0;
//
//	// vmarz?: pick based on light power?
//	int lightIndex = 0;
//	if (1 < lights.size())
//	{
//		float sample = getRandomUniformFloat(&lightPrd.randomState);
//		lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
//	}
//
//	const Light light = lights[lightIndex];
//	const float inverseLightPickPdf = lights.size();
//
//	float3 rayOrigin;
//	float3 rayDirection;
//	float emissionPdfW;
//	float directPdfW;
//	float cosAtLight;
//	lightPrd.throughput = lightEmit(light, lightPrd.randomState, rayOrigin, rayDirection, emissionPdfW, directPdfW, cosAtLight);
//	// vmarz?: do something similar as done for photon, emit towards scene when light far from scene?
//	// check if photons normally missing the scene accounted for?
//	
//	// Set init data
//	emissionPdfW *= inverseLightPickPdf;
//	directPdfW *= inverseLightPickPdf;
//
//	lightPrd.throughput /= emissionPdfW;
//	//lightPrd.isFinite = isDelta.isFinite ... vmarz?
//
//	lightPrd.dVCM = Mis(directPdfW / emissionPdfW);
//
//	// e.g. if not delta ligth
//	if (!light.isDelta)
//	{
//		const float usedCosLight = light.isFinite ? cosAtLight : 1.f;
//		lightPrd.dVC = Mis(usedCosLight / emissionPdfW);
//	}
//
//	lightPrd.dVM = lightPrd.dVC * misVcWeightFactor;
//
//	// Trace
//	Ray lightRay = Ray(rayOrigin, rayDirection, RayType::LIGHT_VCM, 0.0001, RT_DEFAULT_MAX );
//	rtTrace( sceneRootObject, lightRay, lightPrd );
//
//	randomStates[launchIndex] = lightPrd.randomState;
//
//#if ENABLE_RENDER_DEBUG_OUTPUT
//	debugPhotonPathLengthBuffer[launchIndex] = lightPrd.depth;
//#endif
//}


rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );
RT_PROGRAM void miss()
{
    printf("Miss %d,%d - Dep %d - done\n", launchIndex.x, launchIndex.y, lightPrd.depth);
    lightPrd.done = 1;
    OPTIX_DEBUG_PRINT(lightPrd.depth, "Light ray missed geometry.\n");
}


// Exception handler program
rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    printf("Exception Light ray!\n");
    rtPrintExceptionDetails();
    lightPrd.throughput = make_float3(0,0,0);
}