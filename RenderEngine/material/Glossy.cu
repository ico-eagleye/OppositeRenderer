#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2014 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 *
 * Contributions: Valdis Vilcans
*/

#define OPTIX_PRINTF_DEF
#define OPTIX_PRINTFI_DEF
#define OPTIX_PRINTFID_DEF
#define OPTIX_PRINTFC_DEF
#define OPTIX_PRINTFCID_DEF

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ppm/Photon.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/store_photon.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/LightVertex.h"
#include "renderer/vcm/vcm.h"
#include "renderer/vcm/mis.h"
#include "renderer/vcm/config_vcm.h"
#include "renderer/BxDF.h"
#include "renderer/BSDF.h"

#define OPTIX_PRINTF_ENABLED 0
#define OPTIX_PRINTFI_ENABLED 0
#define OPTIX_PRINTFID_ENABLED 0
#define OPTIX_PRINTFC_ENABLED 0
#define OPTIX_PRINTFCID_ENABLED 0

using namespace optix;

rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_STOCHASTIC_HASH
rtDeclareVariable(uint3, photonsGridSize, , );
rtDeclareVariable(float3, photonsWorldOrigo, ,);
rtDeclareVariable(float, photonsGridCellSize, ,);
rtDeclareVariable(unsigned int, photonsSize,,);
rtBuffer<unsigned int, 1> photonsHashTableCount;
#endif

rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, exponent, , );

// TODO Implement coorectly PT and PPM here, now behaves as if it was Diffuse

// Radiance Program
RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.attenuation *= Kd;
    radiancePrd.normal = worldShadingNormal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
    radiancePrd.depth++; 
    if(radiancePrd.flags & PRD_PATH_TRACING)
    {
        float2 sample = getRandomUniformFloat2(&radiancePrd.randomState);
        radiancePrd.randomNewDirection = sampleUnitHemisphereCos(worldShadingNormal, sample);
    }
}


// Photon Program
RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
    {
        Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
        STORE_PHOTON(photon);
    }

    photonPrd.power *= Kd;
    OPTIX_PRINTFID(launchIndex, photonPrd.depth, "Hit Diffuse P(%.2f %.2f %.2f) RT=%d\n", hitPoint.x, hitPoint.y, hitPoint.z, ray.ray_type);
    photonPrd.weight *= fmaxf(Kd);

    // Use russian roulette sampling from depth X to limit the length of the path

    if( photonPrd.depth >= PHOTON_TRACING_RR_START_DEPTH)
    {
        float probContinue = favgf(Kd);
        float probSample = getRandomUniformFloat(&photonPrd.randomState);
        if(probSample >= probContinue )
        {
            return;
        }
        photonPrd.power /= probContinue;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001)
    {
        return;
    }

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU
    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;
#endif

    newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.0001 );
    rtTrace(sceneRootObject, newRay, photonPrd);
}




////////////////////////////////////////////////////////////////////////////////////////////////////
// Vertex Connection and Merging

rtDeclareVariable(Camera,     camera, , );
rtDeclareVariable(float2,     pixelSizeFactor, , );
rtDeclareVariable(SubpathPRD, subpathPrd, rtPayload, );
rtDeclareVariable(uint,       lightVertexCountEstimatePass, , );
rtDeclareVariable(uint,       maxPathLen, , );

rtBuffer<LightVertex>  lightVertexBuffer;
rtBuffer<uint>         lightVertexBufferIndexBuffer; // single element buffer with index for lightVertexBuffer
rtBuffer<uint, 2>      lightSubpathLengthBuffer;

rtDeclareVariable(int, lightVertexBufferId, , );            // rtBufferId<LightVertex>
rtDeclareVariable(int, lightVertexBufferIndexBufferId, , ); // rtBufferId<uint>
rtDeclareVariable(int, lightSubpathVertexCountBufferId, , );// rtBufferId<uint, 2>
rtDeclareVariable(int, outputBufferId, , );                 // rtBufferId<float3, 2>

#if !VCM_UNIFORM_VERTEX_SAMPLING
rtBuffer<uint, 3>       lightSubpathVertexIndexBuffer;
rtDeclareVariable(int,  lightSubpathVertexIndexBufferId, , ); // rtBufferId<uint, 3>
#else
rtDeclareVariable(float, vertexPickPdf, , );                // used for uniform vertex sampling
#endif

rtDeclareVariable(uint,  lightSubpathCount, , );
rtDeclareVariable(float, misVcWeightFactor, , ); // 1/etaVCM
rtDeclareVariable(float, misVmWeightFactor, , ); // etaVCM


 // Light subpath program
RT_PROGRAM void vcmClosestHitLight()
{
    float3 worldGeometricNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometricNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;      
    
    rtBufferId<float3, 2>      _outputBufferId                  = rtBufferId<float3, 2>(outputBufferId);
    rtBufferId<LightVertex, 1> _lightVertexBufferId             = rtBufferId<LightVertex, 1>(lightVertexBufferId);
    rtBufferId<uint, 1>        _lightVertexBufferIndexBufferId  = rtBufferId<uint, 1>(lightVertexBufferIndexBufferId);
    rtBufferId<uint, 1>        _lightSubpathVertexCountBufferId = rtBufferId<uint, 1>(lightSubpathVertexCountBufferId);
#if !VCM_UNIFORM_VERTEX_SAMPLING
    rtBufferId<uint, 2>        _lightSubpathVertexIndexBufferId = rtBufferId<uint, 2>(lightSubpathVertexIndexBufferId);
#endif

    // use geometric normals, shading normals require additional handling due non-symetry for adjoint/reverse bsdfs
    // see [Veach PhD section 5.3]
    VcmBSDF lightBsdf = VcmBSDF(worldGeometricNormal, -ray.direction, true);
    Lambertian lamb(Kd);
    lightBsdf.AddBxDF(&lamb);
    Phong phong(Ks, exponent);
    lightBsdf.AddBxDF(&phong);
    
    const Phong* ph = reinterpret_cast<const Phong*>(lightBsdf.bxdfAt(1));
    OPTIX_PRINTFID(launchIndex, subpathPrd.depth, "Hit C -        phong Kr  % 14f % 14f % 14f\n",  ph->_reflectance.x, ph->_reflectance.y, ph->_reflectance.z);
    
    const Lambertian* la = reinterpret_cast<const Lambertian*>(lightBsdf.bxdfAt(0));
    OPTIX_PRINTFID(launchIndex, subpathPrd.depth, "Hit C -   lambertian Kd     % 14f % 14f % 14f\n", la->_reflectance.x, la->_reflectance.y, la->_reflectance.z);

    lightHit(sceneRootObject, subpathPrd, hitPoint, worldGeometricNormal, lightBsdf, ray.direction, tHit, maxPathLen,
             lightVertexCountEstimatePass, lightSubpathCount, misVcWeightFactor, misVmWeightFactor,
             camera, pixelSizeFactor,
             _outputBufferId, _lightVertexBufferId, _lightVertexBufferIndexBufferId, _lightSubpathVertexCountBufferId,
#if !VCM_UNIFORM_VERTEX_SAMPLING
             _lightSubpathVertexIndexBufferId
#else
             &vertexPickPdf
#endif
        );
}


//rtDeclareVariable(uint, vcmNumlightVertexConnections, , );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );
rtDeclareVariable(float,  averageLightSubpathLength, , );
rtDeclareVariable(int,    lightsBufferId, , );                 // rtBufferId<uint, 1>

 // Camra subpath program
RT_PROGRAM void vcmClosestHitCamera()
{
    float3 worldGeometricNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometricNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    // use geometric normals, shading normals require additional handling due non-symetry for adjoint/reverse bsdfs
    // see [Veach PhD section 5.3]
    VcmBSDF cameraBsdf = VcmBSDF(worldGeometricNormal, -ray.direction, false);
    Lambertian lamb(Kd);
    cameraBsdf.AddBxDF(&lamb);
    Phong phong(Ks, exponent);
    cameraBsdf.AddBxDF(&phong);    

    //OPTIX_PRINTFID(launchIndex, subpathPrd.depth, "Hit C - incident Kr     % 14f % 14f % 14f\n", Kr.x, Kr.y, Kr.z);

    rtBufferId<Light, 1>       _lightsBufferId                  = rtBufferId<Light, 1>(lightsBufferId);
    rtBufferId<LightVertex, 1> _lightVertexBufferId             = rtBufferId<LightVertex, 1>(lightVertexBufferId);
    rtBufferId<uint, 1>        _lightVertexBufferIndexBufferId  = rtBufferId<uint, 1>(lightVertexBufferIndexBufferId);
    rtBufferId<uint, 1>        _lightSubpathVertexCountBufferId = rtBufferId<uint, 1>(lightSubpathVertexCountBufferId);
#if !VCM_UNIFORM_VERTEX_SAMPLING
    rtBufferId<uint, 2>        _lightSubpathVertexIndexBufferId = rtBufferId<uint, 2>(lightSubpathVertexIndexBufferId);
#endif

    cameraHit(sceneRootObject, sceneBoundingSphere, subpathPrd, hitPoint, worldGeometricNormal, cameraBsdf, ray.direction, tHit, maxPathLen,
              lightSubpathCount, misVcWeightFactor, misVmWeightFactor, 
              _lightsBufferId, _lightVertexBufferId, _lightVertexBufferIndexBufferId, _lightSubpathVertexCountBufferId,
#if !VCM_UNIFORM_VERTEX_SAMPLING
              _lightSubpathVertexIndexBufferId
#else
             averageLightSubpathLength,
             &vertexPickPdf
#endif
        );
}