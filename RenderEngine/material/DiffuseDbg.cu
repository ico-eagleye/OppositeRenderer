#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ppm/Photon.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/store_photon.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/PathVertex.h"

using namespace optix;

rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(SubpathPRD, lightPrd, rtPayload, );
rtBuffer<uint, 2> lightVertexCountBuffer;

RT_PROGRAM void closestHitLightDbg()
{
	lightPrd.depth++;    
	if (0.5f < getRandomUniformFloat(&lightPrd.randomState))
	{
		lightPrd.done = 1;
		return;
	}

	//float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
	float3 hitPoint = ray.origin + tHit*ray.direction;
	lightPrd.origin = hitPoint;
	lightPrd.direction = -ray.direction;
	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - point %f %f %f\n", hitPoint.x, hitPoint.y, hitPoint.z);

	//float hitCosTheta = dot(worldShadingNormal, -ray.direction);
	//if (hitCosTheta < 0) return;
	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - cos theta %f \n", hitCosTheta);

	// store path vertex
	//lightVertexCountBuffer[launchIndex] = lightPrd.depth;
	
	// Russian Roulette
	//float contProb = luminanceCIE(Kd);
	//float rrSample = getRandomUniformFloat(&lightPrd.randomState);    
	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - cont %f RR %f \n", contProb, rrSample);
	//if (0.5f < rrSample)
	//{
	//	lightPrd.done = 1;
	//	return;
	//}

	// New dir
	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - new dir %f %f %f\n", lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);	
	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - new org %f %f %f\n", lightPrd.origin.x, lightPrd.origin.y, lightPrd.origin.z);

	// Doesn't crash if code below uncommented
	//if (lightPrd.depth == 1)
	//{
	//    lightPrd.done = 1;
	//    return;
	//}
}


// Create ONB from normalaized vector
static __device__ __inline__ void createONB( 
	const optix::float3& n, optix::float3& U, optix::float3& V)
{
  using namespace optix;

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( dot(U, U) < 1.e-3f )
	  U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( n, U );
}


float3 __device__ __inline__ sampleHemisphereCosOptix(float3 normal, float2 rnd)
{
	float3 p;
	cosine_sample_hemisphere(rnd.x, rnd.y, p);
	float3 v1, v2;
	createONB(normal, v1, v2);
	return v1 * p.x + v2 * p.y + normal * p.z;  
}


RT_PROGRAM void closestHitLightDbgRC()
{
	lightPrd.depth++;

	float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
	float3 hitPoint = ray.origin + tHit*ray.direction;

	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - rayDir %f %f %f\n", ray.direction.x, ray.direction.y, ray.direction.z);
	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - point %f %f %f\n", hitPoint.x, hitPoint.y, hitPoint.z);
	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - normal %f %f %f\n", worldShadingNormal.x, worldShadingNormal.y, worldShadingNormal.z);

	float hitCosTheta = dot(worldShadingNormal, -ray.direction);
	if (hitCosTheta < 0) return;
	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - cos theta %f \n", hitCosTheta);

	// store path vertex
	lightVertexCountBuffer[launchIndex] = lightPrd.depth;
	
	// Russian Roulette
	float contProb = luminanceCIE(Kd);
	float rrSample = getRandomUniformFloat(&lightPrd.randomState);    
	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - cont %f RR %f \n", contProb, rrSample);
	if (contProb < rrSample)
	{
		lightPrd.done = 1;
		return;
	}
	lightPrd.throughput /= contProb;

	float2 bsdfSample = getRandomUniformFloat2(&lightPrd.randomState);
	float3 dir = sampleUnitHemisphereCos(worldShadingNormal, bsdfSample);
	dir = sampleHemisphereCosOptix(worldShadingNormal, bsdfSample);
	//OPTIX_DEBUG_PRINT(lightPrd.depth, "Hit - samp dir %f %f %f len %f\n", dir.x, dir.y, dir.z, sqrtf(dot(dir, dir)));	
	//lightPrd.direction = normalize(dir);
 //   lightPrd.direction = normalize(4*worldShadingNormal + ray.direction);

	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - new dir %f %f %f\n", lightPrd.direction.x, lightPrd.direction.y, lightPrd.direction.z);	
	dir = normalize(dir);

	Ray newRay = make_Ray(hitPoint, dir, RayType::LIGHT_VCM, RAY_LEN_MIN, RT_DEFAULT_MAX);
	rtTrace( sceneRootObject, newRay, lightPrd );

	//OPTIX_DEBUG_PRINT(lightPrd.depth, " Hit - new org %f %f %f\n", lightPrd.origin.x, lightPrd.origin.y, lightPrd.origin.z);

	// Doesn't crash if code below uncommented
	//if (lightPrd.depth == 2)
	//{
	//    lightPrd.done = 1;
	//    return;
	//}
}