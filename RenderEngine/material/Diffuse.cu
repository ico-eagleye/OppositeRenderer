#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

//#define OPTIX_PRINTFID_DISABLE
//#define OPTIX_PRINTFI_DISABLE
//#define OPTIX_PRINTFIALL_DISABLE

#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ppm/Photon.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/store_photon.h"
#include "renderer/vcm/SubpathPRD.h"
#include "renderer/vcm/LightVertex.h"
#include "renderer/vcm/vcm.h"
#include "renderer/vcm/config_vcm.h"
#include "material/BxDF.h"
#include "material/BSDF.h"

using namespace optix;

rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(float3, Kd, , );

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_STOCHASTIC_HASH
rtDeclareVariable(uint3, photonsGridSize, , );
rtDeclareVariable(float3, photonsWorldOrigo, ,);
rtDeclareVariable(float, photonsGridCellSize, ,);
rtDeclareVariable(unsigned int, photonsSize,,);
rtBuffer<unsigned int, 1> photonsHashTableCount;
#endif


/*
// Radiance Program
*/
RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.attenuation *= Kd;
    radiancePrd.normal = worldShadingNormal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
    radiancePrd.depth++; // vmarz: using for debugging (was already defined in struct)
    if(radiancePrd.flags & PRD_PATH_TRACING)
    {
        float2 sample = getRandomUniformFloat2(&radiancePrd.randomState);
        radiancePrd.randomNewDirection = sampleUnitHemisphereCos(worldShadingNormal, sample);
    }
}

/*
// Photon Program
*/
RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
    {
        Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
        STORE_PHOTON(photon);
    }

    photonPrd.power *= Kd;
    OPTIX_PRINTFI(photonPrd.depth, "Hit Diffuse P(%.2f %.2f %.2f) RT=%d\n", hitPoint.x, hitPoint.y, hitPoint.z, ray.ray_type);
    photonPrd.weight *= fmaxf(Kd);

    // Use russian roulette sampling from depth X to limit the length of the path

    if( photonPrd.depth >= PHOTON_TRACING_RR_START_DEPTH)
    {
        float probContinue = favgf(Kd);
        float probSample = getRandomUniformFloat(&photonPrd.randomState);
        if(probSample >= probContinue )
        {
            return;
        }
        photonPrd.power /= probContinue;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001)
    {
        return;
    }

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU
    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;
#endif

    newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.0001 );
    rtTrace(sceneRootObject, newRay, photonPrd);
}




rtDeclareVariable(SubpathPRD, subpathPrd, rtPayload, );
rtDeclareVariable(uint, lightVertexCountEstimatePass, , );
rtBuffer<uint, 2> lightSubpathLengthBuffer;
rtBuffer<LightVertex> lightVertexBuffer;
rtBuffer<uint> lightVertexBufferIndexBuffer; // single element buffer with index for lightVertexBuffer

#if !VCM_UNIFORM_VERTEX_SAMPLING
rtBuffer<uint, 3> lightSubpathVertexIndexBuffer;
rtDeclareVariable(uint, lightSubpathMaxLen, , );
#endif

rtDeclareVariable(float, vertexPickPdf, , );

rtDeclareVariable(float, misVcWeightFactor, , ); // 1/etaVCM
rtDeclareVariable(float, misVmWeightFactor, , ); // etaVCM

//__noinline__ // seems to cause above error
// "_rtContextCompile" caught exception: Assertion failed: "insn->isMove() || insn->isLoad() || insn->isAdd()", [5639172]
__device__ __inline__ void setVcmBSDF(VcmBSDF &bsdf, float3 & aWorldNormal, float3 & aWorldHitDir)
{
    Lambertian lambertian = Lambertian(Kd);
    //OPTIX_PRINTF("setVcmBSDF - Lambertian._reflectance %f %f %f addr 0x%X\n", 
    //    lambertian._reflectance.x, lambertian._reflectance.y, lambertian._reflectance.z, 
    //    (optix::optix_size_t)&lambertian._reflectance);
    bsdf = VcmBSDF(aWorldNormal, aWorldHitDir);
    bsdf.AddBxDF(&lambertian);
}


 // Light subpath program
RT_PROGRAM void closestHitLight()
{
    subpathPrd.depth++;

    // vmarz TODO make sure shading normals used correctly
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - incident dir W  % 14f % 14f % 14f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - hitPoint        % 14f % 14f % 14f\n", hitPoint.x, hitPoint.y, hitPoint.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - normal W        % 14f % 14f % 14f\n", worldShadingNormal.x, worldShadingNormal.y, worldShadingNormal.z);

    // vmarz TODO infinite lights need attitional handling
    float cosThetaIn = dot(worldShadingNormal, -ray.direction);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - cos theta %f \n", cosThetaIn);
    if (cosThetaIn < EPS_COSINE) // reject if cos too low
    {
        subpathPrd.done = 1;
        return;
    }   

    updateMisTermsOnHit(subpathPrd, cosThetaIn, tHit);

    LightVertex lightVertex;
    lightVertex.launchIndex = subpathPrd.launchIndex;
    lightVertex.hitPoint = hitPoint;
    lightVertex.throughput = subpathPrd.throughput;
    lightVertex.pathLen = subpathPrd.depth;
    lightVertex.dVCM = subpathPrd.dVCM;
    lightVertex.dVC = subpathPrd.dVC;
    lightVertex.dVM = subpathPrd.dVM;
#if VCM_UNIFORM_VERTEX_SAMPLING
    lightVertex.dVC = subpathPrd.dVC_unif_vert;
    // There is no dVC_unif_vert in LightVertex since vertices are used only for connection between each other,
    // and do not affect connection to camera/light source and dVC is not present in weight equation for VM.
    // equations in [tech. rep. (38-47)]
#endif
    setVcmBSDF(lightVertex.bsdf, worldShadingNormal, -ray.direction);

    DifferentialGeometry dg = lightVertex.bsdf.differentialGeometry();
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - frame vectors b % 14f % 14f % 14f\n", dg.bitangent.x, dg.bitangent.y, dg.bitangent.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L -               t % 14f % 14f % 14f\n", dg.tangent.x, dg.tangent.y, dg.tangent.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L -               n % 14f % 14f % 14f\n", dg.normal.x, dg.normal.y, dg.normal.z);
    float3 dirFix = lightVertex.bsdf.localDirFix();
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - dir fix local   % 14f % 14f % 14f\n", dirFix.x, dirFix.y, dirFix.z);

    // store path vertex
    if (!lightVertexCountEstimatePass) // vmarz: store flag in PRD ?
    {
#if !VCM_UNIFORM_VERTEX_SAMPLING
        if (subpathPrd.depth == lightSubpathMaxLen)
        {
            OPTIX_PRINTFIALL(subpathPrd.depth, "Hit L - Light path reached MAX LENGTH \n");
            subpathPrd.done = 1;
            return;
        }
#endif
        uint vertIdx = atomicAdd(&lightVertexBufferIndexBuffer[0], 1u);
        OPTIX_PRINTFI(subpathPrd.depth, "Hit L - Vert.throuhput  % 14f % 14f % 14f\n", 
            lightVertex.throughput.x, lightVertex.throughput.y, lightVertex.throughput.z);
        lightVertexBuffer[vertIdx] = lightVertex;

#if !VCM_UNIFORM_VERTEX_SAMPLING
        uint3 pathVertIdx = make_uint3(launchIndex, subpathPrd.depth-1);
        lightSubpathVertexIndexBuffer[pathVertIdx] = vertIdx;
#endif
    }

    // vmarz TODO connect to camera
    // vmarz TODO check max path length
    
    // Russian Roulette
    float contProb =  lightVertex.bsdf.continuationProb(); //luminanceCIE(Kd); // vmarz TODO precompute
    float rrSample = getRandomUniformFloat(&subpathPrd.randomState);    
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - continue sample % 14f             RR % 14f \n", contProb, rrSample);
    if (contProb < rrSample)
    {
        subpathPrd.done = 1;
        return;
    }

    // TODO use BSDF class
    // next event
    float3 bsdfFactor = Kd * M_1_PIf;
    float bsdfDirPdfW;
    float cosThetaOut;
    float2 bsdfSample = getRandomUniformFloat2(&subpathPrd.randomState);
    subpathPrd.direction = sampleUnitHemisphereCos(worldShadingNormal, bsdfSample, &bsdfDirPdfW, &cosThetaOut);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - new dir World   % 14f % 14f % 14f\n",
        subpathPrd.direction.x, subpathPrd.direction.y, subpathPrd.direction.z);

    float bsdfRevPdfW = cosThetaIn * M_1_PIf;
    bsdfDirPdfW *= contProb;
    bsdfRevPdfW *= contProb;
    updateMisTermsOnScatter(subpathPrd, cosThetaOut, bsdfDirPdfW, bsdfRevPdfW, misVcWeightFactor, misVmWeightFactor, &vertexPickPdf);

    OPTIX_PRINTFI(subpathPrd.depth, "Hit L -      bsdfFactor % 14f % 14f % 14f \n", bsdfFactor.x, bsdfFactor.y, bsdfFactor.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - prd.througput1  % 14f % 14f % 14f \n", 
        subpathPrd.throughput.x, subpathPrd.throughput.y, subpathPrd.throughput.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - th=(cosThetaOut % 14f /  bsdfDirPdfW % 14f ) * througput * bsdfactor \n",
        cosThetaOut, bsdfDirPdfW);
    
    // f * cosTheta / f_pdf
    subpathPrd.throughput *= bsdfFactor * (cosThetaOut / bsdfDirPdfW); 
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - prd.througput2  % 14f % 14f % 14f \n", 
        subpathPrd.throughput.x, subpathPrd.throughput.y, subpathPrd.throughput.z);

    subpathPrd.origin = hitPoint;
    OPTIX_PRINTFI(subpathPrd.depth, "Hit L - new origin      % 14f % 14f % 14f\n\n", 
        subpathPrd.origin.x, subpathPrd.origin.y, subpathPrd.origin.z);
}



__inline__
__device__ int isOccluded(optix::float3 point, optix::float3 direction, float tMax)
{
    ShadowPRD shadowPrd;
    shadowPrd.attenuation = 1.0f;
    Ray occlusionRay(point, direction, RayType::SHADOW, EPS_RAY, tMax - 2.f*EPS_RAY);
    rtTrace(sceneRootObject, occlusionRay, shadowPrd);
    return shadowPrd.attenuation == 0.f;
}




// Connects vertices and return contribution
__device__ float3 connectVertices(LightVertex & aLightVertex, VcmBSDF & aCameraBsdf, SubpathPRD & aCameraPrd,
                                  optix::float3 & aCameraHitpoint, const float const * aVertexPickPdf = NULL)
{
    OPTIX_PRINTFI(aCameraPrd.depth, "connectVertices(): \n");
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -  cameraHitPoint % 14f % 14f % 14f\n",
        aCameraHitpoint.x, aCameraHitpoint.y, aCameraHitpoint.z);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - --> vertex      % 14f % 14f % 14f        pathLen % 14d            id %3d %3d \n",
        aLightVertex.hitPoint.x, aLightVertex.hitPoint.y, aLightVertex.hitPoint.z, 
        aLightVertex.pathLen, aLightVertex.launchIndex.x, aLightVertex.launchIndex.y);

    // Get connection
    float3 direction = aLightVertex.hitPoint - aCameraHitpoint;
    float dist2      = dot(direction, direction);
    float distance   = sqrt(dist2);
    direction       /= distance;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -             dir % 14f % 14f % 14f           dist % 14f\n",
      direction.x, direction.y, direction.z, distance);

    // Evaluate BSDF at camera vertex
    float cameraCosTheta, cameraBsdfDirPdfW, cameraBsdfRevPdfW;
    const float3 cameraBsdfFactor = aCameraBsdf.vcmF(direction, cameraCosTheta, &cameraBsdfDirPdfW, &cameraBsdfRevPdfW, 
        &aCameraPrd.launchIndex);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -  cameraCosTheta 14f \n", cameraCosTheta);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -  cameraBsdfFact % 14f % 14f % 14f\n", 
        cameraBsdfFactor.x, cameraBsdfFactor.y, cameraBsdfFactor.z);

    if (isZero(cameraBsdfFactor))
    {
        OPTIX_PRINTFI(aCameraPrd.depth, "conn  - SKIP connect Camera BSDF zero \n");
        return;
    }

    // Add camera continuation probability (for russian roulette)
    const float cameraCont = aCameraBsdf.continuationProb();
    cameraBsdfDirPdfW *= cameraCont;
    cameraBsdfRevPdfW *= cameraCont;

    // Evaluate BSDF at light vertex
    float lightCosTheta, lightBsdfDirPdfW, lightBsdfRevPdfW;
    const float3 lightBsdfFactor = aLightVertex.bsdf.vcmF(-direction, lightCosTheta, &lightBsdfDirPdfW, &lightBsdfRevPdfW,
        &aCameraPrd.launchIndex);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -   lightCosTheta % 14f \n", lightCosTheta);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -   lightBsdfFact % 14f % 14f % 14f\n", lightBsdfFactor.x, lightBsdfFactor.y, lightBsdfFactor.z);

    // Geometry term
    const float geometryTerm = lightCosTheta * cameraCosTheta / dist2;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -    geometryTerm % 14f         dist2 % 14f\n", geometryTerm, dist2);

    if (geometryTerm < 0.f)
        return;

    // Convert solid angle pdfs to area pdfs
    const float cameraBsdfDirPdfA = PdfWtoA(cameraBsdfDirPdfW, distance, cameraCosTheta);
    const float lightBsdfDirPdfA = PdfWtoA(lightBsdfDirPdfW, distance, lightCosTheta);

    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - camBsdfDirPdfA = (camBsdfDirPdfW *       cosLight) / sqr (      distance) \n");
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - % 14f = (% 14f * % 14f) / sqr (% 14f) \n",
        cameraBsdfDirPdfA, cameraBsdfDirPdfW, cameraCosTheta, distance);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - lgtBsdfDirPdfA = (lgtBsdfDirPdfW *      cosCamera) / sqr (      distance) \n");
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - % 14f = (% 14f * % 14f) / sqr (% 14f) \n",
        lightBsdfDirPdfA, lightBsdfDirPdfW, lightCosTheta, distance);

    // aVertPickPdf is set only when unform vertex sampling used (connecting to all paths)
    float invVertPickPdf = aVertexPickPdf ? (1.f / *aVertexPickPdf) : 1.f;
    float aCameraPrd_dVC = aCameraPrd.dVC;
#if VCM_UNIFORM_VERTEX_SAMPLING
    aCameraPrd_dVC = aCameraPrd.dVC_unif_vert;
    // There is no dVC_unif_vert in LightVertex since vertices are used only for connection between each other,
    // and do not affect connection to camera/light source and dVC is not present in weight equation for VM.
    // equations in [tech. rep. (38-47)]
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -  invVertPickPdf % 14f \n", invVertPickPdf);
#endif

    // Partial light sub-path MIS weight [tech. rep. (40)]
    const float wLight = vcmMis(cameraBsdfDirPdfA) * 
        ( misVmWeightFactor * invVertPickPdf + aLightVertex.dVCM + aLightVertex.dVC * vcmMis(lightBsdfRevPdfW) );
    // lightBsdfRevPdfW is Reverse with respect to light path, e.g. in eye path progression 
    // dirrection (note same arrow dirs in formula)
    // note (40) and (41) uses light subpath Y and camera subpath z;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - LightVertex dVC % 14e            dVM % 14e           dVCM % 14e\n",
        aLightVertex.dVC, aLightVertex.dVM, aLightVertex.dVCM);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -         wLight = camBsdfDirPdfA * (VmWeightFactor +     light.dVCM +      light.dVC * lgtBsdfRevPdfW) \n");
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - % 14f = % 14f * (% 14f + % 14e + % 14f * % 14f) \n", 
            wLight, cameraBsdfDirPdfA, misVmWeightFactor, aLightVertex.dVCM, aLightVertex.dVC, lightBsdfRevPdfW);

    // Partial eye sub-path MIS weight [tech. rep. (41)]
    const float wCamera = vcmMis(lightBsdfDirPdfA) * 
        ( misVmWeightFactor * invVertPickPdf + aCameraPrd.dVCM + aCameraPrd_dVC * vcmMis(cameraBsdfRevPdfW) );
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - Camera      dVC % 14e            dVM % 14e           dVCM % 14e\n",
        aCameraPrd_dVC, aCameraPrd.dVM, aCameraPrd.dVCM);    
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -        wCamera = lgtBsdfDirPdfA * (VmWeightFactor +    camera.dVCM +     camera.dVC * camBsdfRevPdfW) \n");
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - % 14f = % 14f * (% 14f + % 14e + % 14f * % 14f) \n", 
            wLight, lightBsdfDirPdfA, misVmWeightFactor, aCameraPrd.dVCM, aCameraPrd.dVC, cameraBsdfRevPdfW);

    // Full path MIS weight [tech. rep. (37)]
    const float misWeight = 1.f / (wLight + 1.f + wCamera);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -       misWeight % 14f         wLight % 14f        wCamera % 14f\n",
        misWeight, wLight, wCamera);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -  Cam througput  % 14f % 14f % 14f\n",
        aCameraPrd.throughput.x, aCameraPrd.throughput.z, aCameraPrd.throughput.y);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - Vert througput  % 14f % 14f % 14f\n",
        aLightVertex.throughput.x, aLightVertex.throughput.z, aLightVertex.throughput.y);

    float3 contrib = geometryTerm * cameraBsdfFactor * lightBsdfFactor * invVertPickPdf;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - noThp unw cntrb % 14f % 14f % 14f \n", contrib.x, contrib.y, contrib.z);
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - noThp wei cntrb = geometryTerm * cameraBsdfFactor * lightBsdfFactor * invVertPickPdf \n");
    contrib *= misWeight;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  - noThp wei cntrb % 14f % 14f % 14f \n", contrib.x, contrib.y, contrib.z);
    contrib *= aCameraPrd.throughput * aLightVertex.throughput;
    OPTIX_PRINTFI(aCameraPrd.depth, "conn  -   Thp wei cntrb % 14f % 14f % 14f \n", contrib.x, contrib.y, contrib.z);

    if (isOccluded(aCameraHitpoint, direction, distance))
    {
        OPTIX_PRINTFI(aCameraPrd.depth, "conn  - OCCLUDED\n");
        return;
    }
    OPTIX_PRINTFI(aCameraPrd.depth, "\n");

    return contrib;
}




rtDeclareVariable(uint, vcmNumlightVertexConnections, , );
rtDeclareVariable(float, averageLightSubpathLength, , );

 // Camra subpath program
RT_PROGRAM void vcmClosestHitCamera()
{
    //OPTIX_PRINTFI(subpathPrd.depth, "CamHit\n");
    subpathPrd.depth++;	

    // vmarz TODO make sure shading normals used correctly
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - incident dir W  % 14f % 14f % 14f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - hitPoint        % 14f % 14f % 14f\n", hitPoint.x, hitPoint.y, hitPoint.z);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - normal W        % 14f % 14f % 14f\n", worldShadingNormal.x, worldShadingNormal.y, worldShadingNormal.z);

    // vmarz TODO infinite lights need attitional handling
    float cosThetaIn = dot(worldShadingNormal, -ray.direction);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - cosThetaIn      % 14f \n", cosThetaIn);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - incident dir W  % 14f % 14f % 14f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    if (cosThetaIn < EPS_COSINE) // reject if cos too low
    {
        subpathPrd.done = 1;
        return;
    }   

    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - cosThetaIn      % 14f         rayLen % 14f\n", cosThetaIn, tHit);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - MIS preUpd  dVC % 14e            dVM % 14e           dVCM % 14e\n",
        subpathPrd.dVC, subpathPrd.dVM, subpathPrd.dVCM);
    updateMisTermsOnHit(subpathPrd, cosThetaIn, tHit);
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - MIS postUpd dVC % 14e            dVM % 14e           dVCM % 14e\n",
        subpathPrd.dVC, subpathPrd.dVM, subpathPrd.dVCM);

    VcmBSDF cameraBsdf;
    setVcmBSDF(cameraBsdf, worldShadingNormal, -ray.direction);
    // TODO connect to light source

    // Connect to ligth vertices // TODO move to func
#if VCM_UNIFORM_VERTEX_SAMPLING
    uint numLightVertices = lightVertexBufferIndexBuffer[0];
    //float vertexPickPdf = float(vcmNumlightVertexConnections) / numLightVertices; // TODO scale by pick prob
    uint numlightVertexConnections = ceilf(averageLightSubpathLength);
    float lastVertConnectProb = averageLightSubpathLength - (uint)averageLightSubpathLength;
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - CONNECT     num % 14u   lastVertProb % 14f \n", 
        numlightVertexConnections, lastVertConnectProb);
    //for (int i = 0; i < vcmNumlightVertexConnections; i++)
    for (int i = 0; i < numlightVertexConnections; i++)
    {
        // For last vertex do russian roulette
        if (i == (numlightVertexConnections - 1))
        {
            float sampleConnect = getRandomUniformFloat(&subpathPrd.randomState);
            if (lastVertConnectProb < sampleConnect)
                break;
        }

        uint vertIdx = numLightVertices * getRandomUniformFloat(&subpathPrd.randomState);
        LightVertex lightVertex = lightVertexBuffer[vertIdx];
        subpathPrd.color += connectVertices(lightVertex, cameraBsdf, subpathPrd, hitPoint, &vertexPickPdf);
    }
#else
    uint lightSubpathLen = lightSubpathLengthBuffer[launchIndex];
    uint3 pathVertIdx = make_uint3(launchIndex, 0u);
    for (int i = 0; i < lightSubpathLen; i++)
    {
        uint vertIdx = lightSubpathVertexIndexBuffer[pathVertIdx];
        LightVertex lightVertex = lightVertexBuffer[vertIdx];
        subpathPrd.color += connectVertices(lightVertex, cameraBsdf, subpathPrd, hitPoint);
        pathVertIdx.z++;
    }
#endif
    
    // vmarz TODO check max path length
    // Russian Roulette
    float contProb =  cameraBsdf.continuationProb();// luminanceCIE(Kd); // vmarz TODO precompute
    float rrSample = getRandomUniformFloat(&subpathPrd.randomState);    
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - continue sample % 14f             RR % 14f \n", contProb, rrSample);
    if (contProb < rrSample)
    {
        subpathPrd.done = 1;
        return;
    }

    // next event
    float3 bsdfFactor = Kd * M_1_PIf;
    float bsdfDirPdfW;
    float cosThetaOut;
    float2 bsdfSample = getRandomUniformFloat2(&subpathPrd.randomState);
    subpathPrd.direction = sampleUnitHemisphereCos(worldShadingNormal, bsdfSample, &bsdfDirPdfW, &cosThetaOut);
    //OPTIX_PRINTFI(subpathPrd.depth, "Hit - new dir %f %f %f\n", subpathPrd.direction.x, subpathPrd.direction.y, subpathPrd.direction.z);

    float bsdfRevPdfW = cosThetaIn * M_1_PIf;
    bsdfDirPdfW *= contProb;
    bsdfRevPdfW *= contProb;
    updateMisTermsOnScatter(subpathPrd, cosThetaOut, bsdfDirPdfW, bsdfRevPdfW, misVcWeightFactor, misVmWeightFactor, &vertexPickPdf);

    // f * cosTheta / f_pdf
    subpathPrd.throughput *= bsdfFactor * (cosThetaOut / bsdfDirPdfW);
    subpathPrd.origin = hitPoint;
    OPTIX_PRINTFI(subpathPrd.depth, "Hit C - new origin     % 14f % 14f % 14f\n", 
      subpathPrd.origin.x, subpathPrd.origin.y, subpathPrd.origin.z);
}